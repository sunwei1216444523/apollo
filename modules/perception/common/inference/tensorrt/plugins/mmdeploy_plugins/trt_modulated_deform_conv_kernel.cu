#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright 2024 The Apollo Authors. All Rights Reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *****************************************************************************/

#include <assert.h>
#include <hip/hip_fp16.h>

#include "modules/perception/common/inference/tensorrt/plugins/mmdeploy_plugins/common_cuda_helper.hpp"
// #include "modulated_deform_conv/modulated_deform_conv_cuda.cu"
#include "modules/perception/common/inference/tensorrt/plugins/mmdeploy_plugins/trt_modulated_deform_conv_kernel.hpp"
#include "modules/perception/common/inference/tensorrt/plugins/mmdeploy_plugins/trt_plugin_helper.hpp"

namespace apollo {
namespace perception {
namespace inference {

template <typename T>
__device__ float mdcn_im2col_bilinear(const T *input, const int data_width, const int height,
                                      const int width, float h, float w) {
  int h_low = floorf(h);
  int w_low = floorf(w);
  int h_high = h_low + 1;
  int w_high = w_low + 1;

  T lh = h - h_low;
  T lw = w - w_low;
  T hh = 1 - lh, hw = 1 - lw;

  T v1 = 0;
  if (h_low >= 0 && w_low >= 0) v1 = input[h_low * data_width + w_low];
  T v2 = 0;
  if (h_low >= 0 && w_high <= width - 1) v2 = input[h_low * data_width + w_high];
  T v3 = 0;
  if (h_high <= height - 1 && w_low >= 0) v3 = input[h_high * data_width + w_low];
  T v4 = 0;
  if (h_high <= height - 1 && w_high <= width - 1) v4 = input[h_high * data_width + w_high];

  T w1 = hh * hw, w2 = hh * lw, w3 = lh * hw, w4 = lh * lw;

  T val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);
  return float(val);
}
template <>
__device__ float mdcn_im2col_bilinear<__half>(const __half *input, const int data_width,
                                              const int height, const int width, float h, float w) {
  int h_low = floorf(h);
  int w_low = floorf(w);
  int h_high = h_low + 1;
  int w_high = w_low + 1;

  float lh = h - h_low;
  float lw = w - w_low;
  float hh = 1 - lh, hw = 1 - lw;

  float v1 = 0;
  if (h_low >= 0 && w_low >= 0) v1 = __half2float(input[h_low * data_width + w_low]);
  float v2 = 0;
  if (h_low >= 0 && w_high <= width - 1) v2 = __half2float(input[h_low * data_width + w_high]);
  float v3 = 0;
  if (h_high <= height - 1 && w_low >= 0) v3 = __half2float(input[h_high * data_width + w_low]);
  float v4 = 0;
  if (h_high <= height - 1 && w_high <= width - 1)
    v4 = __half2float(input[h_high * data_width + w_high]);

  float w1 = hh * hw, w2 = hh * lw, w3 = lh * hw, w4 = lh * lw;

  float val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);
  return val;
}

template <typename T>
__global__ void modulated_deformable_im2col_gpu_kernel(
    const int n, const T *data_im, const T *data_offset, const T *data_mask, const int height,
    const int width, const int kernel_h, const int kernel_w, const int pad_h, const int pad_w,
    const int stride_h, const int stride_w, const int dilation_h, const int dilation_w,
    const int channel_per_deformable_group, const int batch_size, const int num_channels,
    const int deformable_group, const int height_col, const int width_col, T *data_col) {
  CUDA_1D_KERNEL_LOOP(index, n) {
    // index index of output matrix
    const int w_col = index % width_col;
    const int h_col = (index / width_col) % height_col;
    const int b_col = (index / width_col / height_col) % batch_size;
    const int c_im = (index / width_col / height_col) / batch_size;
    const int c_col = c_im * kernel_h * kernel_w;

    // compute deformable group index
    const int deformable_group_index = c_im / channel_per_deformable_group;

    const int h_in = h_col * stride_h - pad_h;
    const int w_in = w_col * stride_w - pad_w;

    T *data_col_ptr =
        data_col + ((c_col * batch_size + b_col) * height_col + h_col) * width_col + w_col;
    const T *data_im_ptr = data_im + (b_col * num_channels + c_im) * height * width;
    const T *data_offset_ptr = data_offset + (b_col * deformable_group + deformable_group_index) *
                                                 2 * kernel_h * kernel_w * height_col * width_col;

    const T *data_mask_ptr = data_mask + (b_col * deformable_group + deformable_group_index) *
                                             kernel_h * kernel_w * height_col * width_col;

    for (int i = 0; i < kernel_h; ++i) {
      for (int j = 0; j < kernel_w; ++j) {
        const int data_offset_h_ptr =
            ((2 * (i * kernel_w + j)) * height_col + h_col) * width_col + w_col;
        const int data_offset_w_ptr =
            ((2 * (i * kernel_w + j) + 1) * height_col + h_col) * width_col + w_col;
        const int data_mask_hw_ptr = ((i * kernel_w + j) * height_col + h_col) * width_col + w_col;
        const T offset_h = data_offset_ptr[data_offset_h_ptr];
        const T offset_w = data_offset_ptr[data_offset_w_ptr];
        const T mask = data_mask_ptr[data_mask_hw_ptr];
        float val = 0.0f;
        const float h_im = h_in + i * dilation_h + (float)offset_h;
        const float w_im = w_in + j * dilation_w + (float)offset_w;
        if (h_im > -1 && w_im > -1 && h_im < height && w_im < width)
          val = mdcn_im2col_bilinear(data_im_ptr, width, height, width, h_im, w_im);
        *data_col_ptr = (T)(val * (float)mask);
        data_col_ptr += batch_size * height_col * width_col;
      }
    }
  }
}

template <typename T>
void trt_modulated_deformable_im2col(const T* data_im_, const T* data_offset_, const T* data_mask_,
                                     const int batch_size, const int channels, const int height_im,
                                     const int width_im, const int height_col, const int width_col,
                                     const int kernel_h, const int kenerl_w, const int pad_h,
                                     const int pad_w, const int stride_h, const int stride_w,
                                     const int dilation_h, const int dilation_w,
                                     const int deformable_group, T* data_col_,
                                     hipStream_t stream) {
  // num_axes should be smaller than block size
  const int channel_per_deformable_group = channels / deformable_group;
  const int num_kernels = channels * batch_size * height_col * width_col;

  modulated_deformable_im2col_gpu_kernel<T>
      <<<GET_BLOCKS(num_kernels), THREADS_PER_BLOCK, 0, stream>>>(
          num_kernels, data_im_, data_offset_, data_mask_, height_im, width_im, kernel_h, kenerl_w,
          pad_h, pad_w, stride_h, stride_w, dilation_h, dilation_w, channel_per_deformable_group,
          batch_size, channels, deformable_group, height_col, width_col, data_col_);

  cudaCheckError();
}

template <typename scalar_t>
__global__ void output_add_bias_kernel(scalar_t* output, const scalar_t* bias, size_t step_batch,
                                       size_t step_channel, size_t n) {
  CUDA_1D_KERNEL_LOOP(index, n) { output[index] += bias[(index % step_batch) / step_channel]; }
}

#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 530)
template <>
__global__ void output_add_bias_kernel<__half>(__half* output, const __half* bias,
                                               size_t step_batch, size_t step_channel, size_t n) {
  CUDA_1D_KERNEL_LOOP(index, n) {
    const __half b = bias[(index % step_batch) / step_channel];
    const __half o = output[index];
    output[index] = __hadd(o, b);
  }
}
#else
template <>
__global__ void output_add_bias_kernel<__half>(__half* output, const __half* bias,
                                               size_t step_batch, size_t step_channel, size_t n) {
  CUDA_1D_KERNEL_LOOP(index, n) {
    const __half b = bias[(index % step_batch) / step_channel];
    const __half o = output[index];
    output[index] = __float2half(__half2float(o) + __half2float(b));
  }
}
#endif

template <typename scalar_t>
static void output_add_bias(scalar_t* output, const scalar_t* bias, size_t batch, size_t channel,
                            size_t height, size_t width, hipStream_t stream) {
  size_t step_channel = height * width;
  size_t step_batch = step_channel * channel;
  size_t n = step_batch * batch;
  output_add_bias_kernel<<<GET_BLOCKS(n), THREADS_PER_BLOCK, 0, stream>>>(output, bias, step_batch,
                                                                          step_channel, n);
}

template <typename scalar_t>
void ModulatedDeformConvForwardCUDAKernelLauncher(
    const scalar_t* input, const scalar_t* weight, const scalar_t* bias, const scalar_t* offset,
    const scalar_t* mask, scalar_t* output, void* workspace, int batch, int channels, int height,
    int width, int channels_out, int kernel_w, int kernel_h, int stride_w, int stride_h, int pad_w,
    int pad_h, int dilation_w, int dilation_h, int group, int deformable_group, int im2col_step,
    hipblasHandle_t cublas_handle, hipStream_t stream) {
  bool with_bias = (bias != nullptr);

  im2col_step = std::min(int(batch), im2col_step);
  assert(batch % im2col_step == 0);

  const int height_out = (height + 2 * pad_h - (dilation_h * (kernel_h - 1) + 1)) / stride_h + 1;
  const int width_out = (width + 2 * pad_w - (dilation_w * (kernel_w - 1) + 1)) / stride_w + 1;

  scalar_t* columns = (scalar_t*)workspace;

  const size_t input_step = channels * height * width;
  const size_t offset_step = deformable_group * kernel_h * kernel_w * 2 * height_out * width_out;
  const size_t mask_step = deformable_group * kernel_h * kernel_w * height_out * width_out;
  const size_t out_step = channels_out * height_out * width_out;
  const size_t out_group_step = out_step / group;
  const size_t col_g_step = channels * kernel_w * kernel_h / group * height_out * width_out;
  const size_t weight_g_step = channels_out / group * channels / group * kernel_h * kernel_w;

  const int m = channels_out / group;
  const int n = height_out * width_out;
  const int k = channels / group * kernel_h * kernel_w;
  scalar_t alpha = 1.;
  scalar_t beta = 0.;

  for (int b = 0; b < batch; b++) {
    const scalar_t* input_start = input + b * input_step;
    const scalar_t* offset_start = offset + b * offset_step;
    const scalar_t* mask_start = mask + b * mask_step;
    trt_modulated_deformable_im2col<scalar_t>(
        input_start, offset_start, mask_start, 1, channels, height, width, height_out, width_out,
        kernel_h, kernel_w, pad_h, pad_w, stride_h, stride_w, dilation_h, dilation_w,
        deformable_group, columns, stream);

    for (int g = 0; g < group; g++) {
      const scalar_t* weight_start = weight + g * weight_g_step;
      scalar_t* col_start = columns + g * col_g_step;
      scalar_t* out_buffer_start = output + b * out_step + g * out_group_step;

      cublasGemmWrap<scalar_t>(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, col_start,
                               n, weight_start, k, &beta, out_buffer_start, n);
      cudaCheckError();
    }
  }

  if (with_bias) {
    output_add_bias<scalar_t>(output, bias, batch, channels_out, height_out, width_out, stream);
  }
}

template void ModulatedDeformConvForwardCUDAKernelLauncher<float>(
    const float* input, const float* weight, const float* bias, const float* offset,
    const float* mask, float* output, void* workspace, int batch, int channels, int height,
    int width, int channels_out, int kernel_w, int kernel_h, int stride_w, int stride_h, int pad_w,
    int pad_h, int dilation_w, int dilation_h, int group, int deformable_group, int im2col_step,
    hipblasHandle_t cublas_handle, hipStream_t stream);

template void ModulatedDeformConvForwardCUDAKernelLauncher<__half>(
    const __half* input, const __half* weight, const __half* bias, const __half* offset,
    const __half* mask, __half* output, void* workspace, int batch, int channels, int height,
    int width, int channels_out, int kernel_w, int kernel_h, int stride_w, int stride_h, int pad_w,
    int pad_h, int dilation_w, int dilation_h, int group, int deformable_group, int im2col_step,
    hipblasHandle_t cublas_handle, hipStream_t stream);

}  // namespace inference
}  // namespace perception
}  // namespace apollo
